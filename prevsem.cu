#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <math.h>

__global__ void exam(char* A, int N)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if ((row == 0 || row == N - 1) || (col == 0 || col == N - 1))
	{
		A[row * N + col] = '!';
	}
	int i;
	int count = 0;
	for (i = 1; i <= row; i++)
	{
		if (row % i == 0)
			count++;
	}
	if (count == 2 && A[row * N + col]=='\0')
	{
		A[row * N + col] = '*';
	}
	else if (count != 2 && A[row * N + col] == '\0')
	{
		A[row * N + col] = '#';
	}
	else
	{
		char e = A[row * N + col];
		if(e=='a'|| e == 'e' || e == 'i' || e == 'o' || e == 'u')
		{
			A[row * N + col] = e - 32;
		}
		else if (e == 'A' || e == 'E' || e == 'I' || e == 'O' || e == 'U')
		{
			A[row * N + col] = e + 32;
		}
	}
	
}

int main()
{
	int N;
	printf("Enter N:");
	scanf("%d", &N);
	int i, j;
	char* A = (char*)malloc(N * N * sizeof(char));
	printf("Enter matrix\n");
	char* temp=(char*)malloc(N * sizeof(char));
	for (i = 0; i < N; i++)
	{
		temp[i] = '\0';
	}
	int k = 0;
	for (i = 0; i < N; i++)
	{
		scanf("%s", temp);
		for (j = 0; j < N; j++)
		{
			A[k++] = temp[j];
		}
	}
	
	/*for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%c ",A[i*N+j]);
		}
		printf("\n");
	}*/
	char* d_A;
	hipMalloc((void**)&d_A, N * N * sizeof(char));

	hipMemcpy(d_A, A, N * N * sizeof(char), hipMemcpyHostToDevice);

	dim3 g(2, 2, 1);
	dim3 b(ceil(N / 2), ceil(N / 2), 1);
	exam << <g, b >> > (d_A,N);
	hipMemcpy(A,d_A, N * N * sizeof(char), hipMemcpyDeviceToHost);
	printf("Result:\n");
	for (i = 0; i < N; i++)
	{
		for (j = 0; j < N; j++)
		{
			printf("%c ",A[i*N+j]);
		}
		printf("\n");
	}

}