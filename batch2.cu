#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"
#include""

__device__ int isfib(int num)
{
    int a = 1, b = 1, isfib = 0;
    while (num >= b)
    {
        if (num == b)
        {
            isfib = 1;
            break;
        }
        else
        {
            int temp = a + b;
            a = b;
            b = temp;
        }
    }
    if (isfib == 1)
        return 1;
    else
        return 0;
}
__device__ int octal(int num)
{
    int octal_num = 0, countval = 1;
    while (num != 0) {

        int remainder = num % 8;
        octal_num += remainder * countval;
        countval = countval * 10;
        num = num / 8;
    }
    return octal_num;
}
__global__ void fun(int* A, int* B, int M, int N, int* totalfibcount)
{
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    if (row == 0 || col == 0 || row == M - 1 || col == N - 1)
    {
        B[(row)*N + col] = octal(A[(row)*N + col]);
        if(isfib(A[(row)*N + col]))
        {
            int a = atomicAdd(totalfibcount, 1);
        }
    }
    else
    {
        int fibcount = 0;
        fibcount += isfib(A[(row)*N + (col - 1)]);
        fibcount += isfib(A[(row)*N + (col + 1)]);
        fibcount += isfib(A[(row - 1) * N + (col)]);
        fibcount += isfib(A[(row + 1) * N + (col)]);
        fibcount += isfib(A[(row - 1) * N + (col - 1)]);
        fibcount += isfib(A[(row + 1) * N + (col - 1)]);
        fibcount += isfib(A[(row - 1) * N + (col + 1)]);
        fibcount += isfib(A[(row + 1) * N + (col + 1)]);
        B[row * N + col] = fibcount;
        if(isfib(A[(row)*N + col]))
        {
            int a = atomicAdd(totalfibcount, 1);
        }
    }
}
int main()
{
    int M = 4, N = 4;
    int A[100], B[100];
    printf("Enter the matrix elements of A: ");
    for (int i = 0; i < 4; i++)
    {
        for (int j = 0; j < 4; j++)
            scanf("%d", &A[i * N + j]);
    }
    printf("A:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%d\t", A[i * N + j]);
        printf("\n");
    }
    int totalfibcount = 0;
    int size = M * N * sizeof(int);
    int* d_A, * d_B, * d_totalfibcount;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_totalfibcount, sizeof(int));
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_totalfibcount, &totalfibcount, sizeof(int), hipMemcpyHostToDevice);
    dim3 dimBlock(N, M, 1);
    dim3 dimGrid(1, 1, 1);
    fun <<< dimGrid, dimBlock >> > (d_A, d_B, M, N, d_totalfibcount);
    hipMemcpy(B, d_B, size, hipMemcpyDeviceToHost);
    hipMemcpy(&totalfibcount, d_totalfibcount, sizeof(int), hipMemcpyDeviceToHost);
    printf("B:\n");
    for (int i = 0; i < M; i++)
    {
        for (int j = 0; j < N; j++)
            printf("%d\t", B[i * N + j]);
        printf("\n");
    }
    printf("Total Fibonacci count: %d\n", totalfibcount);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_totalfibcount);
}